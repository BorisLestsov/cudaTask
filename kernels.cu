#include <hip/hip_runtime.h>


__global__
void solution_ker(double* A, int M, int N, int K){
   int i = threadIdx.x;
   int j = threadIdx.y;
   int k = threadIdx.z;
   int ind = (i*N+j)*K + k; 
   if (i == 0 || j == M-1 || j == 0 || j == N-1 || k == 0 | k == K-1)   {
       A[ind] = 10.*i ; //finish this
   } else {
       A[ind] = 0.0;
   }
}

#define a(i,j,k) a[((i)*nn+(j))*kk+(k)]
#define b(i,j,k) b[((i)*nn+(j))*kk+(k)]

__global__
void jac_comp(double* a, double* b, int mm, int nn, int kk, int BLOCKSIZE){
    int bl_i = blockIdx.x;
    int bl_j = blockIdx.y;
    int bl_k = blockIdx.z;
    int th_i = threadIdx.x;
    int th_j = threadIdx.y;
    int th_k = threadIdx.z;
    int i = BLOCKSIZE*bl_i + th_i + 1;
    int j = BLOCKSIZE*bl_j + th_j + 1;
    int k = BLOCKSIZE*bl_k + th_k + 1;
    if (i >= mm-1 || j >= nn-1 || k >= kk-1)
        return;

    b(i,j,k) = (a(i-1, j ,k) + a(i+1, j, k)
             +  a(i, j-1, k) + a(i, j+1, k)
             +  a(i, j, k-1) + a(i, j, k+1)) / 6.;
   //b(i,j,k) = 1.0;
}

__global__
void jac_eps(double* a, double* b, int mm, int nn, int kk){
   int i = threadIdx.x+1;
   int j = threadIdx.y+1;
   int k = threadIdx.z+1;
   b(i,j,k) = (a(i-1, j ,k) + a(i+1, j, k)
            +  a(i, j-1, k) + a(i, j+1, k)
            +  a(i, j, k-1) + a(i, j, k+1)) / 6.;
}


