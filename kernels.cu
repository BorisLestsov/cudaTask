#include <hip/hip_runtime.h>
#include <float.h>


__global__
void solution_ker(double* A, int M, int N, int K){
   int i = threadIdx.x;
   int j = threadIdx.y;
   int k = threadIdx.z;
   int ind = (i*N+j)*K + k; 
   if (i == 0 || j == M-1 || j == 0 || j == N-1 || k == 0 | k == K-1)   {
       A[ind] = 10.*i ; //finish this
   } else {
       A[ind] = 0.0;
   }
}

#define a(i,j,k) a[((i)*nn+(j))*kk+(k)]
#define b(i,j,k) b[((i)*nn+(j))*kk+(k)]
#define res(i,j,k) res[((i)*nn+(j))*kk+(k)]

__global__
void jac_comp(double* a, double* b, int mm, int nn, int kk, int BLOCKSIZE){
    int bl_i = blockIdx.x;
    int bl_j = blockIdx.y;
    int bl_k = blockIdx.z;
    int th_i = threadIdx.x;
    int th_j = threadIdx.y;
    int th_k = threadIdx.z;
    int i = BLOCKSIZE*bl_i + th_i + 1;
    int j = BLOCKSIZE*bl_j + th_j + 1;
    int k = BLOCKSIZE*bl_k + th_k + 1;
    if (i >= mm-1 || j >= nn-1 || k >= kk-1)
        return;

    b(i,j,k) = (a(i-1, j ,k) + a(i+1, j, k)
             +  a(i, j-1, k) + a(i, j+1, k)
             +  a(i, j, k-1) + a(i, j, k+1)) / 6.;
    //b(i,j,k) = 1.0;
}

__global__
void no_bound_memcpy(double* a, double* b, int mm, int nn, int kk, int BLOCKSIZE, double value){
    int bl_i = blockIdx.x;
    int bl_j = blockIdx.y;
    int bl_k = blockIdx.z;
    int th_i = threadIdx.x;
    int th_j = threadIdx.y;
    int th_k = threadIdx.z;
    int i = BLOCKSIZE*bl_i + th_i + 1;
    int j = BLOCKSIZE*bl_j + th_j + 1;
    int k = BLOCKSIZE*bl_k + th_k + 1;
    if (i >= mm-1 || j >= nn-1 || k >= kk-1)
        return;

    a(i,j,k) = b(i,j,k);
}

__global__
void fill(double* a, double value, int mm, int nn, int kk, int BLOCKSIZE){
    int bl_i = blockIdx.x;
    int bl_j = blockIdx.y;
    int bl_k = blockIdx.z;
    int th_i = threadIdx.x;
    int th_j = threadIdx.y;
    int th_k = threadIdx.z;
    int i = BLOCKSIZE*bl_i + th_i;
    int j = BLOCKSIZE*bl_j + th_j;
    int k = BLOCKSIZE*bl_k + th_k;

    a(i,j,k) = value;
}


__global__
void jac_diff(double* res, double* a, double* b, int mm, int nn, int kk, int BLOCKSIZE){
    int bl_i = blockIdx.x;
    int bl_j = blockIdx.y;
    int bl_k = blockIdx.z;
    int th_i = threadIdx.x;
    int th_j = threadIdx.y;
    int th_k = threadIdx.z;
    int i = BLOCKSIZE*bl_i + th_i + 1;
    int j = BLOCKSIZE*bl_j + th_j + 1;
    int k = BLOCKSIZE*bl_k + th_k + 1;
    if (i >= mm-1 || j >= nn-1 || k >= kk-1)
        return;

    res(i,j,k) = fabs(a(i,j,k)-b(i,j,k));
}


__device__
float atomicMaxf(float* address, float val)
{
    int *address_as_int =(int*)address;
    int old = *address_as_int, assumed;
    while (val > __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
                __float_as_int(val));
    }
    return __int_as_float(old);
}


__global__
void jac_max(double* d_array, float* d_max, int elements)
{
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int gid = (blockDim.x * blockIdx.x) + tid;
    shared[tid] = -FLT_MAX; 

    while (gid < elements) {
        shared[tid] = max(shared[tid], d_array[gid]);
        gid += gridDim.x*blockDim.x;
    }
    __syncthreads();
    gid = (blockDim.x * blockIdx.x) + tid;
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s && gid < elements)
            shared[tid] = max(shared[tid], shared[tid + s]);
        __syncthreads();
    }

    if (tid == 0)
        atomicMaxf(d_max, __double2float_rn(shared[0]));
}


__global__
void max_reduce_no_bounds(double* a, int mm, int nn, int kk, int BLOCKSIZE){
    

}

