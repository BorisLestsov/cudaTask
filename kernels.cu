#include <hip/hip_runtime.h>


__global__
void solution_ker(double* A, int M, int N, int K){
   int i = threadIdx.x;
   int j = threadIdx.y;
   int k = threadIdx.z;
   int ind = (i*N+j)*K + k; 
   if (i == 0 || j == M-1 || j == 0 || j == N-1 || k == 0 | k == K-1)   {
       A[ind] = 10.*i ; //finish this
   } else {
       A[ind] = 0.0;
   }
}

#define a(i,j,k) a[((i)*nn+(j))*kk+(k)]
#define b(i,j,k) b[((i)*nn+(j))*kk+(k)]

__global__
void jac_comp(double* a, double* b, int mm, int nn, int kk){
   int i = threadIdx.x+1;
   int j = threadIdx.y+1;
   int k = threadIdx.z+1;
   //b(i,j,k) = (a(i-1, j ,k) + a(i+1, j, k)
   //         +  a(i, j-1, k) + a(i, j+1, k)
   //         +  a(i, j, k-1) + a(i, j, k+1)) / 6.;
   b(i,j,k) = 1.0;
}

__global__
void jac_eps(double* a, double* b, int mm, int nn, int kk){
   int i = threadIdx.x+1;
   int j = threadIdx.y+1;
   int k = threadIdx.z+1;
   b(i,j,k) = (a(i-1, j ,k) + a(i+1, j, k)
            +  a(i, j-1, k) + a(i, j+1, k)
            +  a(i, j, k-1) + a(i, j, k+1)) / 6.;
}


