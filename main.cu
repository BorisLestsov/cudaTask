#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>

#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include "mpi.h"

#define  Max(a,b) ((a)>(b)?(a):(b))

FILE *in;
int TRACE = 1;
int i, j, k, it;
double EPS;
int     M, N, K, ITMAX;
double  MAXEPS = 0.1;
double time0;

double *A;
#define A(i,j,k) A[((i)*N+(j))*K+(k)]

double solution(int i, int j, int k)
{
    double x = 10.*i / (M - 1), y = 10.*j / (N - 1), z = 10.*k / (K - 1);
    return 2.*x*x - y*y - z*z;
    /*    return x+y+z; */
}

double jac(double *a, int mm, int nn, int kk, int itmax, double maxeps);

int main(int an, char **as)
{

    in = fopen("data3.in", "r");
    if (in == NULL) { printf("Can not open 'data3.in' "); exit(1); }
    i = fscanf(in, "%d %d %d %d %d", &M, &N, &K, &ITMAX, &TRACE);
    if (i < 4) 
    {
        printf("Wrong 'data3.in' (M N K ITMAX TRACE)");
        exit(2);
    }

    A = (double*) malloc(M*N*K*sizeof(double));

    for (i = 0; i <= M - 1; i++)
        for (j = 0; j <= N - 1; j++)
            for (k = 0; k <= K - 1; k++)
            {
                if (i == 0 || i == M - 1 || j == 0 || j == N - 1 || k == 0 || k == K - 1)
                    A(i, j, k) = solution(i, j, k);
                else 
                    A(i, j, k) = 0.;
            }


    printf("%dx%dx%d x %d\t<", M, N, K, ITMAX);
    time0 = 0.;
    EPS = jac(A, M, N, K, ITMAX, MAXEPS);   
    
    printf("%3.1f>\teps=%.4g ", time0, EPS);

    if (TRACE)
    {
        EPS = 0.;

        for (i = 0; i <= M - 1; i++)
            for (j = 0; j <= N - 1; j++)
                for (k = 0; k <= K - 1; k++)
                    EPS = Max(fabs(A(i, j, k) - solution(i, j, k)), EPS);
        printf("delta=%.4g\n", EPS);
    }

    free(A);
    return 0;
}

#define a(i,j,k) a[((i)*nn+(j))*kk+(k)]
#define b(i,j,k) b[((i)*nn+(j))*kk+(k)]

#define BLOCKSIZE 8
#define REDUCE_THREADS 128

double jac(double *a, int mm, int nn, int kk, int itmax, double maxeps)
{
    gpuErrchk(hipSetDevice(0));
    
    double *b, *b_d, *a_d, *d_buf;
    double eps;

    float flt_min = FLT_MIN;
    float* d_eps;
    gpuErrchk(hipMalloc(&d_eps, sizeof(float)));


    b = (double*) malloc(mm*nn*kk*sizeof(double));

    gpuErrchk(hipMalloc(&b_d, mm*nn*kk*sizeof(double)));
    gpuErrchk(hipMalloc(&a_d, mm*nn*kk*sizeof(double)));
    gpuErrchk(hipMalloc(&d_buf, mm*nn*kk*sizeof(double)));

    int mm_dim = (mm-2)/BLOCKSIZE + ((mm-2)%BLOCKSIZE!=0);
    int nn_dim = (nn-2)/BLOCKSIZE + ((nn-2)%BLOCKSIZE!=0);
    int kk_dim = (kk-2)/BLOCKSIZE + ((kk-2)%BLOCKSIZE!=0);
    dim3 blockGrid  = dim3(mm_dim, nn_dim, kk_dim);
    dim3 threadGrid = dim3(BLOCKSIZE, BLOCKSIZE, BLOCKSIZE);

    int mm_dim1 = (mm)/BLOCKSIZE + ((mm)%BLOCKSIZE!=0);
    int nn_dim1 = (nn)/BLOCKSIZE + ((nn)%BLOCKSIZE!=0);
    int kk_dim1 = (kk)/BLOCKSIZE + ((kk)%BLOCKSIZE!=0);
    dim3 blockGrid1  = dim3(mm_dim1, nn_dim1, kk_dim1);
    dim3 threadGrid1 = dim3(BLOCKSIZE, BLOCKSIZE, BLOCKSIZE);


    gpuErrchk(hipMemcpy(a_d, a, mm*nn*kk*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(b_d, a_d, mm*nn*kk*sizeof(double), hipMemcpyDeviceToDevice));
    fill<<<blockGrid1, threadGrid1>>>(d_buf, -DBL_MAX, mm, nn, kk, BLOCKSIZE);    
    
    for (it = 1; it <= itmax - 1; it++)
    {
        jac_comp<<<blockGrid, threadGrid>>>(a_d, b_d, mm, nn, kk, BLOCKSIZE);
        //gpuErrchk(hipMemcpy(b, b_d, mm*nn*kk*sizeof(double), hipMemcpyDeviceToHost));
        
        
        jac_diff<<<blockGrid, threadGrid>>>(d_buf, a_d, b_d, mm, nn, kk, BLOCKSIZE);
        
        //for (i = 0; i < mm; ++i)
        //for (j = 0; j < nn; ++j)
        //for (k = 0; k < kk; ++k)
        //printf("%d %d %d %f \n", i, j, k, b(i,j,k));

        float epsf;
        gpuErrchk(hipMemcpy(d_eps, &flt_min, sizeof(float), hipMemcpyHostToDevice));
        int blocks = (mm*nn*kk)/REDUCE_THREADS + ((mm*nn*kk)%REDUCE_THREADS!=0);
        max_reduce<<<blocks, REDUCE_THREADS, REDUCE_THREADS*sizeof(double)>>>(d_buf, d_eps, mm*nn*kk);
        
        gpuErrchk(hipMemcpy(&epsf, d_eps, sizeof(float), hipMemcpyDeviceToHost));
        eps = (double) epsf;        
        
        gpuErrchk(hipMemcpy(a_d, b_d, mm*nn*kk*sizeof(double), hipMemcpyDeviceToDevice));
        //gpuErrchk(hipGetLastError());
        if (TRACE && it%TRACE == 0)
            printf("\nIT=%d eps=%.4g\t", it, eps);
        if (eps < maxeps) 
            break;
    }
    free(b);
    return eps;
}
